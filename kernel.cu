﻿#include "hip/hip_runtime.h"
#include ""

#include "VectorFunc.h"    // Include custom array functions (e.g., CreateRandomArray)
#include "CountingSort.cuh"  // Include Counting Sort implementation
#include "RadixSort.h"     // Include Radix Sort implementation

#include <iostream>
#include <chrono>  // For measuring execution time
// Define the number of elements of the integer array
int const NumberOfElements = 20; // Example: 450 million elements

// Function to measure the time and performance of Counting Sort
void CountingSortAnalysis(int* randomList, int lowerBound, int upperBound)
{
    // Start the timer to measure execution time
    auto start = std::chrono::high_resolution_clock::now();

    // Perform Counting Sort on the random list
    countingSort(upperBound, NumberOfElements, randomList);

    // Stop the timer after sorting is complete
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate the time duration in milliseconds
    std::chrono::duration<double, std::milli> duration = end - start;

    // Output the time taken to sort using Counting Sort
    std::cout << "Time taken to sort the list using Counting sort: " << duration.count() << " milliseconds" << std::endl;
}

// Function to measure the time and performance of Radix Sort
void RadixSortAnalysis(int* randomList, int lowerBound, int upperBound)
{
    // Start the timer to measure execution time
    auto start = std::chrono::high_resolution_clock::now();

    // Perform Radix Sort on the random list
    RadixSort(NumberOfElements, randomList);

    // Stop the timer after sorting is complete
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate the time duration in milliseconds
    std::chrono::duration<double, std::milli> duration = end - start;

    // Output the time taken to sort using Radix Sort
    std::cout << "Time taken to sort the list using Radix sort: " << duration.count() << " milliseconds" << std::endl;
}

#define threadsperblock 1024

// Function to measure the time and performance of Radix Sort
void CumulativeSumAnalysis(int* randomList, int lowerBound, int upperBound)
{
    size_t bytes = sizeof(int) * NumberOfElements;

    // Device vector pointers
    int* d_randomList;

    // Allocate device memory (GPU)
    hipMalloc(&d_randomList, bytes);
    int blocksPerGrid = (NumberOfElements + (threadsperblock - 1)) / threadsperblock;




    // Copy to device
    hipMemcpy(d_randomList, randomList, bytes, hipMemcpyHostToDevice);
    //printArray(randomList, NumberOfElements);
    // Start the timer to measure execution time
    auto start = std::chrono::high_resolution_clock::now();


    // Call kernel
    CumulativeSum << <blocksPerGrid, threadsperblock >> > (d_randomList, NumberOfElements);

    // Stop the timer after sorting is complete
    auto end = std::chrono::high_resolution_clock::now();


    // copy data from device memory to host memory (CPU to  GPU)
    hipMemcpy(randomList, d_randomList, bytes, hipMemcpyDeviceToHost);


    //printArray(randomList, NumberOfElements);
    // Calculate the time duration in milliseconds
    std::chrono::duration<double, std::milli> duration = end - start;

    // Output the time taken to sort using Radix Sort
    std::cout << "Time taken for cumulative sum: " << duration.count() << " milliseconds" << std::endl;
}





int main()
{
    // Define the range for the random numbers (e.g., values between 0 and 99)
    int lowerBound = 1;
    int upperBound = 99;

    // Generate a random list of integers of size NumberOfElements
    int* h_randomList = CreateRandomArray(NumberOfElements, lowerBound, upperBound);
    
    CumulativeSumAnalysis(h_randomList, lowerBound, upperBound);
    

    h_randomList = CreateRandomArray(NumberOfElements, lowerBound, upperBound);
    // Allocation size for all vectors
   CountingSortAnalysis(h_randomList, lowerBound, upperBound);


    //h_randomList = CreateRandomArray(NumberOfElements, lowerBound, upperBound);
    //RadixSortAnalysis(h_randomList, lowerBound, upperBound);


    /*
    printArray(randomList, NumberOfElements);
    // Analyze the performance of Counting Sort
    CountingSortAnalysis(randomList, lowerBound, upperBound);
    printArray(randomList, NumberOfElements);
    // Analyze the performance of Radix Sort
    //RadixSortAnalysis(randomList, lowerBound, upperBound)   
    */
;

    // Free the dynamically allocated randomList array

    return 0;
}
