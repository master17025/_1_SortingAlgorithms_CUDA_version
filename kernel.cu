#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "VectorFunc.h"    // Include custom array functions (e.g., CreateRandomArray)
#include "CountingSort.cuh"  // Include Counting Sort implementation
#include "RadixSort.h"     // Include Radix Sort implementation

#include <iostream>
#include <chrono>  // For measuring execution time

#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>

// Define the number of elements of the integer array
int const NumberOfElements = 1e7; // Example: 450 million elements
#define threadsperblock 1024


// CUDA Kernel to generate random numbers
__global__ void GenerateRandomArrayKernel(int* d_array, int lowerBound, int upperBound, unsigned long seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < NumberOfElements) {
        hiprandState state;
        hiprand_init(seed, tid, 0, &state);

        float randomValue = hiprand_uniform(&state); // Generate random float in range (0, 1]
        d_array[tid] = lowerBound + (int)((upperBound - lowerBound + 1) * randomValue);
    }
}

// Function to generate a random array using CUDA
int* CreateRandomArray(int NumberOfElements, int lowerBound, int upperBound) {
    int* d_array;
    int* h_array = new int[NumberOfElements];

    // Allocate device memory
    hipMalloc(&d_array, sizeof(int) * NumberOfElements);

    // Configure the kernel
    int blocksPerGrid = (NumberOfElements + threadsperblock - 1) / threadsperblock;
    unsigned long seed = time(0); // Random seed

    // Launch the kernel
    GenerateRandomArrayKernel << <blocksPerGrid, threadsperblock >> > (d_array, lowerBound, upperBound, seed);

    // Copy the generated random numbers back to host memory
    hipMemcpy(h_array, d_array, sizeof(int) * NumberOfElements, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_array);

    return h_array;
}

// Function to measure the time and performance of Counting Sort
void CountingSortAnalysis(int* randomList, int lowerBound, int upperBound) {
    auto start = std::chrono::high_resolution_clock::now();
    countingSort(upperBound, NumberOfElements, randomList);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> duration = end - start;
    std::cout << "Time taken to sort the list using Counting sort: " << duration.count() << " milliseconds" << std::endl;
}

// Function to measure the time and performance of Radix Sort
void RadixSortAnalysis(int* randomList, int lowerBound, int upperBound) {
    auto start = std::chrono::high_resolution_clock::now();
    RadixSort(NumberOfElements, randomList);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> duration = end - start;
    std::cout << "Time taken to sort the list using Radix sort: " << duration.count() << " milliseconds" << std::endl;
}

// Function to measure the time and performance of Radix Sort
void CumulativeSumAnalysis(int* randomList, int lowerBound, int upperBound)
{
    size_t bytes = sizeof(int) * NumberOfElements;

    // Device vector pointers
    int* d_randomList;

    // Allocate device memory (GPU)
    hipMalloc(&d_randomList, bytes);
    int blocksPerGrid = (NumberOfElements + (threadsperblock - 1)) / threadsperblock;



    // Copy to device
    hipMemcpy(d_randomList, randomList, bytes, hipMemcpyHostToDevice);
    //printArray(randomList, NumberOfElements);


        // Start the timer to measure execution time
    auto start = std::chrono::high_resolution_clock::now();

    // Call kernel
    CumulativeSum << <blocksPerGrid, threadsperblock >> > (d_randomList, NumberOfElements);

    // Stop the timer after sorting is complete
    auto end = std::chrono::high_resolution_clock::now();

    // copy data from device memory to host memory (CPU to  GPU)
    hipMemcpy(randomList, d_randomList, bytes, hipMemcpyDeviceToHost);



    //printArray(randomList, NumberOfElements);
    // Calculate the time duration in milliseconds
    std::chrono::duration<double, std::milli> duration = end - start;

    // Output the time taken to sort using Radix Sort
    std::cout << "Time taken for cumulative sum: " << duration.count() << " milliseconds" << std::endl;
}



int main() {
    int lowerBound = 1;
    int upperBound = 99;

    // Generate random array on GPU
    int* h_randomList = CreateRandomArray(NumberOfElements, lowerBound, upperBound);

    // Perform analysis
    CumulativeSumAnalysis(h_randomList, lowerBound, upperBound);
    CountingSortAnalysis(h_randomList, lowerBound, upperBound);
    RadixSortAnalysis(h_randomList, lowerBound, upperBound);

    // Free host memory
    delete[] h_randomList;

    return 0;
}
