#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "VectorFunc.h"    // Include custom array functions (e.g., CreateRandomArray)
#include "CountingSort.cuh"  // Include Counting Sort implementation
#include "RadixSort.h"     // Include Radix Sort implementation

#include <iostream>
#include <chrono>  // For measuring execution time

#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>

// Define the number of elements of the integer array
int const NumberOfElements = 1e7; // Example: 450 million elements
#define threadsperblock 1024

// Kernel to initialize CURAND states
__global__ void InitCurandStates(hiprandState* states, unsigned long seed, int NumberOfElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NumberOfElements) {
        hiprand_init(seed, tid, 0, &states[tid]);
    }
}

// Kernel to generate random numbers using pre-initialized states
__global__ void GenerateRandomArrayKernel(int* d_array, hiprandState* states, int lowerBound, int upperBound, int NumberOfElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NumberOfElements) {
        hiprandState localState = states[tid];  // Use pre-initialized state
        float randomValue = hiprand_uniform(&localState); // Generate random float in range (0, 1]
        d_array[tid] = lowerBound + (int)((upperBound - lowerBound + 1) * randomValue);
        states[tid] = localState; // Save updated state
    }
}

// Function to generate a random array using CUDA
int* CreateRandomArray(int NumberOfElements, int lowerBound, int upperBound) {
    int* d_array;
    int* h_array = new int[NumberOfElements];
    hiprandState* d_states;

    // Allocate device memory
    hipMalloc(&d_array, sizeof(int) * NumberOfElements);
    hipMalloc(&d_states, sizeof(hiprandState) * NumberOfElements);

    // Configure kernel
    int blocksPerGrid = (NumberOfElements + threadsperblock - 1) / threadsperblock;
    unsigned long seed = time(0);

    // Initialize CURAND states
    InitCurandStates << <blocksPerGrid, threadsperblock >> > (d_states, seed, NumberOfElements);

    // Generate random numbers using pre-initialized states
    GenerateRandomArrayKernel << <blocksPerGrid, threadsperblock >> > (d_array, d_states, lowerBound, upperBound, NumberOfElements);

    // Copy the generated random numbers back to host memory
    hipMemcpy(h_array, d_array, sizeof(int) * NumberOfElements, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_array);
    hipFree(d_states);

    return h_array;
}

// Function to measure the time and performance of Counting Sort
void CountingSortAnalysis(int* randomList, int lowerBound, int upperBound) {
    auto start = std::chrono::high_resolution_clock::now();
    countingSort(upperBound, NumberOfElements, randomList);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> duration = end - start;
    std::cout << "Time taken to sort the list using Counting sort: " << duration.count() << " milliseconds" << std::endl;
}

// Function to measure the time and performance of Radix Sort
void RadixSortAnalysis(int* randomList, int lowerBound, int upperBound) {
    auto start = std::chrono::high_resolution_clock::now();
    RadixSort(NumberOfElements, randomList);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> duration = end - start;
    std::cout << "Time taken to sort the list using Radix sort: " << duration.count() << " milliseconds" << std::endl;
}

// Function to measure the time and performance of Radix Sort
void CumulativeSumAnalysis(int* randomList, int lowerBound, int upperBound)
{
    size_t bytes = sizeof(int) * NumberOfElements;

    // Device vector pointers
    int* d_randomList;

    // Allocate device memory (GPU)
    hipMalloc(&d_randomList, bytes);
    int blocksPerGrid = (NumberOfElements + (threadsperblock - 1)) / threadsperblock;



    // Copy to device
    hipMemcpy(d_randomList, randomList, bytes, hipMemcpyHostToDevice);
    //printArray(randomList, NumberOfElements);


        // Start the timer to measure execution time
    auto start = std::chrono::high_resolution_clock::now();

    // Call kernel
    CumulativeSum << <blocksPerGrid, threadsperblock >> > (d_randomList, NumberOfElements);

    // Stop the timer after sorting is complete
    auto end = std::chrono::high_resolution_clock::now();

    // copy data from device memory to host memory (CPU to  GPU)
    hipMemcpy(randomList, d_randomList, bytes, hipMemcpyDeviceToHost);



    //printArray(randomList, NumberOfElements);
    // Calculate the time duration in milliseconds
    std::chrono::duration<double, std::milli> duration = end - start;

    // Output the time taken to sort using Radix Sort
    std::cout << "Time taken for cumulative sum: " << duration.count() << " milliseconds" << std::endl;
}



int main() {
    int lowerBound = 1;
    int upperBound = 9;

    // Generate random array on GPU
    int* h_randomList = CreateRandomArray(NumberOfElements, lowerBound, upperBound);

    // Perform analysis
    //CumulativeSumAnalysis(h_randomList, lowerBound, upperBound);
    //CountingSortAnalysis(h_randomList, lowerBound, upperBound);
    //RadixSortAnalysis(h_randomList, lowerBound, upperBound);

    // Free host memory
    delete[] h_randomList;

    return 0;
}
