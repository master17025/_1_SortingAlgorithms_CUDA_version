#include "hip/hip_runtime.h"
#include"RandomGeneration.cuh"

// Kernel to initialize CURAND states
__global__ void InitCurandStates(hiprandState* states, unsigned long seed, int NumberOfElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NumberOfElements) {
        hiprand_init(seed, tid, 0, &states[tid]);
    }
}

// Kernel to generate random numbers using pre-initialized states
__global__ void GenerateRandomArrayKernel(int* d_array, hiprandState* states, int lowerBound, int upperBound, int NumberOfElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NumberOfElements) {
        hiprandState localState = states[tid];  // Use pre-initialized state
        float randomValue = hiprand_uniform(&localState); // Generate random float in range (0, 1]
        d_array[tid] = lowerBound + (int)((upperBound - lowerBound + 1) * randomValue);
        states[tid] = localState; // Save updated state
    }
}

// Function to generate a random array using CUDA
int* CreateRandomArray(int NumberOfElements, int lowerBound, int upperBound) {
    int* d_array;
    int* h_array = new int[NumberOfElements];
    hiprandState* d_states;

    // Allocate device memory
    hipMalloc(&d_array, sizeof(int) * NumberOfElements);
    hipMalloc(&d_states, sizeof(hiprandState) * NumberOfElements);

    // Configure kernel
    int blocksPerGrid = (NumberOfElements + threadsperblock - 1) / threadsperblock;
    unsigned long seed = time(0);

    // Initialize CURAND states
    InitCurandStates << <blocksPerGrid, threadsperblock >> > (d_states, seed, NumberOfElements);

    // Generate random numbers using pre-initialized states
    GenerateRandomArrayKernel << <blocksPerGrid, threadsperblock >> > (d_array, d_states, lowerBound, upperBound, NumberOfElements);

    // Copy the generated random numbers back to host memory
    hipMemcpy(h_array, d_array, sizeof(int) * NumberOfElements, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_array);
    hipFree(d_states);

    return h_array;
}